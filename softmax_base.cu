#include "hip/hip_runtime.h"
// ==========================================================
// softmax_kernel.cu — Multiple CUDA kernel variants
// ==========================================================

#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <float.h>
#include <chrono>

#define DEBUG_MODE  0  // <-- change to 1 to re-enable debug prints

#if DEBUG_MODE
    #define DBG_PRINT(...)   printf(__VA_ARGS__)
#else
    #define DBG_PRINT(...)
#endif


#define MAX_BLOCK_DIM_SIZE 65535
#ifndef MIN
#define MIN(x,y) ((x)<(y)?(x):(y))
#endif

#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

// ------------------------------------------------------------------
// Utility: warp-level reduction for fast sum using shuffle
// ------------------------------------------------------------------
__inline__ __device__ float warpReduceSum(float val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__inline__ __device__ float warpReduceMax(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
        val = fmaxf(val, __shfl_down_sync(FULL_MASK, val, offset));
    return val;
}


// ================================================================
// Variant 0 — Naive softmax (global memory only)
// Each thread computes exp(x[i]) and accumulates partial sums
// ================================================================
__global__ void softmax_naive_kernel(const float* x, float* y,
                                     int N, float* partialSum) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    float local_sum = 0.f;

    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        float val = expf(x[i]);  // No numerical stabilization
        y[i] = val;
        local_sum += val;
    }

    // Store partial sum into shared memory
    __shared__ float sdata[256];
    sdata[tid] = local_sum;
    __syncthreads();

    // Reduce within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
        if (tid < s) sdata[tid] += sdata[tid + s];
    __syncthreads();

    if (tid == 0)
        partialSum[blockIdx.x] = sdata[0];
}

// ================================================================
// Variant 1 — Shared-memory reduction with better coalescing
// ================================================================
__global__ void softmax_shared_kernel(const float* x, float* y,
                                      int N, float* partialSum) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    float val = (i < N) ? expf(x[i]) : 0.f;
    y[i] = val;
    sdata[tid] = val;
    __syncthreads();

    // Block reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
        if (tid < s) sdata[tid] += sdata[tid + s];
    __syncthreads();

    if (tid == 0)
        partialSum[blockIdx.x] = sdata[0];
}

// ================================================================
// Variant 2 — Warp-shuffle reduction (fastest)
// ================================================================
__global__ void softmax_warp_kernel(const float* x, float* y,
                                    int N, float* partialSum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float local_sum = 0.f;

    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        float val = expf(x[i]);
        y[i] = val;
        local_sum += val;
    }

    float sum = warpReduceSum(local_sum);
    __shared__ float warpSum[32];
    if ((threadIdx.x & 31) == 0) warpSum[threadIdx.x / 32] = sum;
    __syncthreads();

    float block_sum = 0.f;
    if (threadIdx.x < 32)
        block_sum = warpReduceSum((threadIdx.x < blockDim.x/32) ? warpSum[threadIdx.x] : 0.f);

    if (threadIdx.x == 0)
        partialSum[blockIdx.x] = block_sum;
}

__global__ void softmax_warp_shared_kernel(const float* __restrict__ x,
                                         float* __restrict__ y,
                                         int N,
                                         float* __restrict__ partialSum) {
    // Shared memory for one float per warp
    __shared__ float warpSums[32];  // enough for up to 1024 threads

    int tid  = threadIdx.x;
    int gid  = blockIdx.x * blockDim.x + tid;

    // --- 1. Coalesced load: one element per thread
    float val = 0.f;
    if (gid < N) {
        val = expf(x[gid]);  // expf is heavy, dominates runtime
        y[gid] = val;
    }

    // --- 2. Warp-level reduction of val across threads in the same warp
    float local_sum = warpReduceSum(val);

    // --- 3. Lane 0 of each warp writes its warp sum to shared memory
    int warpId = tid / warpSize;
    if ((tid & 31) == 0)
        warpSums[warpId] = local_sum;
    __syncthreads();

    // --- 4. Warp 0 reduces all warp sums
    float block_sum = 0.f;
    if (warpId == 0) {
        float warp_val = (tid < blockDim.x / warpSize) ? warpSums[tid] : 0.f;
        block_sum = warpReduceSum(warp_val);
    }

    // --- 5. Store per-block sum
    if (tid == 0)
        partialSum[blockIdx.x] = block_sum;
}

__global__ void softmax_warp_shared_double_kernel(const float* x, float* y, int N, float* partialSum) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 2 + tid;

    float mySum = 0.f;
    if (i < N) mySum = expf(x[i]);
    if (i + blockDim.x < N) mySum += expf(x[i + blockDim.x]);
    y[i] = mySum;
    sdata[tid] = mySum;
    __syncthreads();

    // Reduction in shared memory
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    // Warp-level reduction (unrolled)
    if (tid < 32) {
        float val = sdata[tid];
        for (int offset = 16; offset > 0; offset /= 2)
            val += __shfl_down_sync(0xffffffff, val, offset);
        if (tid == 0)
            partialSum[blockIdx.x] = val;
    }
}

__global__ void softmax_warp_vectorized_kernel(const float *__restrict__ x,
                                                     float *__restrict__ y,
                                                     int N,
                                                     float *__restrict__ partialSum) {
    extern __shared__ float warpBuf[];
    int tid   = threadIdx.x;
    int lane  = tid % WARP_SIZE;
    int warpId = tid / WARP_SIZE;

    int vecIndex = blockIdx.x * blockDim.x + tid;
    int baseIdx  = vecIndex * 4;
    const float4 *x4 = reinterpret_cast<const float4 *>(x);

    float4 val4 = {0.f, 0.f, 0.f, 0.f};
    float local_sum = 0.f;

    int numVec = N / 4;

    // vectorized load (or tail-safe scalar)
    if (vecIndex < numVec) {
        val4 = x4[vecIndex];
    } else if (baseIdx < N) {
        val4.x = (baseIdx + 0 < N) ? x[baseIdx + 0] : -FLT_MAX;
        val4.y = (baseIdx + 1 < N) ? x[baseIdx + 1] : -FLT_MAX;
        val4.z = (baseIdx + 2 < N) ? x[baseIdx + 2] : -FLT_MAX;
        val4.w = (baseIdx + 3 < N) ? x[baseIdx + 3] : -FLT_MAX;
    }

    // compute exp(x)
    float4 exp4;
    exp4.x = expf(val4.x);
    exp4.y = expf(val4.y);
    exp4.z = expf(val4.z);
    exp4.w = expf(val4.w);
    local_sum = exp4.x + exp4.y + exp4.z + exp4.w;

    // store exp(x) directly
    if (baseIdx < N)
        reinterpret_cast<float4 *>(y)[vecIndex] = exp4;

    // reduce for sum
    float warp_sum = warpReduceSum(local_sum);
    if (lane == 0) warpBuf[warpId] = warp_sum;
    __syncthreads();

    float block_sum = 0.f;
    if (warpId == 0) {
        float v = (tid < blockDim.x / WARP_SIZE) ? warpBuf[lane] : 0.f;
        float tmp = warpReduceSum(v);
        if (lane == 0) warpBuf[0] = tmp;
    }
    __syncthreads();
    block_sum = warpBuf[0];

    if (tid == 0)
        partialSum[blockIdx.x] = block_sum;
}


// ================================================================
// Normalization kernel (same for all variants)
// ================================================================
__global__ void normalize_kernel(float* y, float *totalSum, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < N; i += gridDim.x * blockDim.x)
        y[i] /= *totalSum;
}

// ================================================================
// Kernel dispatcher
// ================================================================
template <class T>
void softmax_launch(int N, int threads, int blocks,
                    int whichKernel, T *d_input, T *d_output,
                    T *d_partial) {
    switch (whichKernel) {
        default:
        case 0:
            softmax_naive_kernel<<<blocks, threads, threads * sizeof(T)>>>(
                d_input, d_output, N, d_partial);
            break;
        case 1:
            softmax_shared_kernel<<<blocks, threads, threads * sizeof(T)>>>(
                d_input, d_output, N, d_partial);
            break;
        case 2:
            softmax_warp_kernel<<<blocks, threads, 0>>>(
                d_input, d_output, N, d_partial);
            break;
        case 3: 
            softmax_warp_shared_kernel<<<blocks, threads, threads * sizeof(T)>>>(
                d_input, d_output, N, d_partial);    
            break;  
        case 4: 
            softmax_warp_shared_double_kernel<<<blocks, threads, threads * sizeof(T)>>>(
                d_input, d_output, N, d_partial);    
            break;  
        case 5: 
            int vecN = (N + 3) / 4;  // number of float4s
            int threads_vec = threads;
            int blocks_vec  = (vecN + threads_vec - 1) / threads_vec;
            softmax_warp_vectorized_kernel<<<blocks_vec, threads_vec, threads_vec * sizeof(T)>>>(
                d_input, d_output, N, d_partial);   

            break;    
    }
}

// // Explicit instantiation
// template void softmax_launch<float>(int, int, int, int, float*, float*, float*);

// ================================================================
// CPU reference softmax (for correctness verification)
// ================================================================
template <class T>
void softmax_cpu(const T* x, T* y, int N) {
    double sum = 0.0;
    for (int i = 0; i < N; ++i) {
        y[i] = exp(x[i]);
        sum += y[i];
    }
    for (int i = 0; i < N; ++i)
        y[i] /= sum;
}

template void softmax_cpu<float>(const float*, float*, int);

// Compute next power of 2 (used to tune thread count)
unsigned int nextPow2(unsigned int x) {
    --x; x |= x >> 1; x |= x >> 2; x |= x >> 4;
    x |= x >> 8; x |= x >> 16; return ++x;
}

// ---------------------------------------------------------------
// Utility: compute blocks/threads configuration (like reduction sample)
// ---------------------------------------------------------------
void  getNumBlocksAndThreads(int kernel, int N, int maxBlocks,
                            int maxThreads, int &blocks, int &threads) {
    hipDeviceProp_t prop; int dev;
    checkCudaErrors(hipGetDevice(&dev));
    checkCudaErrors(hipGetDeviceProperties(&prop, dev));

    threads = (N < maxThreads) ? nextPow2(N) : maxThreads;
    blocks  = (N + threads - 1) / threads;

    if (blocks > prop.maxGridSize[0]) {
        blocks = prop.maxGridSize[0];
    }
}

// ================================================================
// Device reduction kernel: reduce array of block partial sums
// ================================================================
__global__ void reduce_partial_sum_kernel(const float* __restrict__ d_partial,
                                          float* __restrict__ d_total,
                                          int numBlocks)
{
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    float val = (idx < numBlocks) ? d_partial[idx] : 0.f;
    sdata[tid] = val;
    __syncthreads();

    // Parallel reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            sdata[tid] += sdata[tid + stride];
        __syncthreads();
    }

    // Write one partial sum per block
    if (tid == 0)
        d_total[blockIdx.x] = sdata[0];
}

// ---------------------------------------------------------------
// Benchmark one variant of softmax for given N
// ---------------------------------------------------------------
float benchmarkSoftmax(int N, int threads, int blocks, int whichKernel,
                       float* d_input, float* d_output, float* d_partial) {
    // --- CUDA event-based timing for GPU-only measurement ---
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // --- Start GPU timing ---
    hipEventRecord(startEvent);

    // 1️⃣ Launch softmax kernel (computes exp(x[i]) and writes to d_output)
    softmax_launch<float>(N, threads, blocks, whichKernel, d_input, d_output, d_partial);
    checkCudaErrors(hipDeviceSynchronize());

    // 2️⃣ Reduce per-block partial sums (entirely on GPU)
    int threads_reduce = 256;
    int blocks_reduce = (blocks + threads_reduce - 1) / threads_reduce;

    reduce_partial_sum_kernel<<<blocks_reduce, threads_reduce,
                                 threads_reduce * sizeof(float)>>>(d_partial, d_partial, blocks);
    checkCudaErrors(hipDeviceSynchronize());

    if (blocks_reduce > 1) {
        reduce_partial_sum_kernel<<<1, threads_reduce,
                                     threads_reduce * sizeof(float)>>>(d_partial, d_partial, blocks_reduce);
        checkCudaErrors(hipDeviceSynchronize());
    }

    // 3️⃣ Normalize output using the final d_partial[0] on GPU
    normalize_kernel<<<blocks, threads>>>(d_output, d_partial, N);
    checkCudaErrors(hipDeviceSynchronize());

    // --- Stop GPU timing ---
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);

    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    return elapsed_ms;
}


// ---------------------------------------------------------------
// Generate Shmoo table across kernel variants and input sizes
// ---------------------------------------------------------------
int main(int argc, char **argv) {
    printf("=== CUDA Softmax Benchmark (no max(x) normalization) ===\n");
    auto t_start = std::chrono::high_resolution_clock::now();
    uint64_t Nmax;
    //int Nmax = 1 << 24;
    int maxThreads = 256;
    int numVariants = 6;
    int maxBlocks = 64;
  
    if (argc > 1) {
        Nmax = atoll(argv[1]);   // user-specified value
        //printf("[INFO] Using custom Nmax = %llu elements\n", (unsigned long long)Nmax);
        Nmax = nextPow2(Nmax) ;
    } else {
        Nmax = 1ULL << 24;       // default (16 million)
       // printf("[INFO] Using default Nmax = %llu elements\n", (unsigned long long)Nmax);
    }

    //int maxBlocks  = std::min(65535, int((Nmax + maxThreads - 1) / maxThreads));
    size_t bytes = Nmax * sizeof(float);
    std::vector<float> h_input(Nmax);
    
    for (int i = 0; i < Nmax; ++i)
        h_input[i] = sinf(i * 0.001f) + (i % 10) * 0.1f;

    // Allocate device memory
    float *d_input, *d_output, *d_partial;
    checkCudaErrors(hipMalloc(&d_input, bytes));
    checkCudaErrors(hipMalloc(&d_output, bytes));
    hipMalloc(&d_partial, (Nmax / maxThreads + 1) * sizeof(float));
    checkCudaErrors(hipMemcpy(d_input, h_input.data(), bytes,
                               hipMemcpyHostToDevice));
    std::vector<float> h_partial(Nmax / maxThreads + 1);

    // CSV header
    printf("Variant");
    for (int n = 1<<10; n <= Nmax; n <<= 1)
        printf(", %d", n);
    printf("\n");

    // Benchmark each variant
    for (int k = 0; k < numVariants; ++k) {
        printf("%d", k);
        for (int n = 1<<10; n <= Nmax; n <<= 1) {
            int threads = 0, blocks = 0;
            getNumBlocksAndThreads(k, n, maxBlocks, maxThreads,
                                   blocks, threads);
            float time_ms = benchmarkSoftmax(n, threads, blocks, k,
                                             d_input, d_output,
                                             d_partial);
            printf(", %.5f", time_ms);
        }
        printf("\n");
    }

    // // ---- Validate correctness for final variant ----
    // int Ntest = std::min<uint64_t>(Nmax, 1ULL << 20);
    // std::vector<float> h_out(Ntest), h_ref(Ntest);
    // checkCudaErrors(hipMemcpy(h_out.data(), d_output,
    //                            Ntest*sizeof(float), hipMemcpyDeviceToHost));
    // softmax_cpu<float>(h_input.data(), h_ref.data(), Ntest);

    // float max_err = 0.f;
    // for (int i=0;i<Ntest;i++)
    //     max_err = fmaxf(max_err, fabsf(h_out[i] - h_ref[i]));
    // printf("\nVerification max abs error: %.6e\n", max_err);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_partial);
    printf("✅ Benchmark completed.\n");
    
    auto t_end = std::chrono::high_resolution_clock::now();
    double total_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    printf("⏱️  Total runtime (end-to-end): %.3f ms\n", total_ms);
    return 0;
}
